
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define  tpb 32

__global__ void bitMask(int* nums,int*len, int* out, int* last, int*bit, int*value){
    int index=threadIdx.x + blockIdx.x*tpb;
    if (index<*len) out[index]=(((nums[index]>>(*bit))%2)==*value);
    if(index==((*len)-1)) *last=out[index];
}

__global__ void exToIn(int* inp, int* out, int*len, int*last){
    int index = threadIdx.x + blockIdx.x*tpb;
    if((index>0)&&(index<*len)){
        out[index-1]=inp[index];
    }
    if(index==((*len)-1)) { out[index]=inp[index]+*last;
    *last=out[index];
    }
}

__global__ void upSweep(int* arr, int* len, int* tLen, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(index>*tLen) return;
    if((((index+1)%(step*2))!=0) || index==0 || ((*len)<=index)) return;
    arr[index]=arr[index]+arr[index-step];
}

__global__ void downSweep(int* arr, int* len, int* tLen, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(2*step==*len) arr[(*len)-1]=0;
    if((((index+1)%(step*2))!=0) || (index==0) || ((*len)<=index)) return;    
    int tmp=arr[index-step];
    arr[index-step]=arr[index];
    arr[index]+=tmp;
}

__global__ void printArr(int* arr,int*len){
    //printf("%d",*len);
    for(int i=0;i<(*len);i++) printf("%d\n",arr[i]);
}

__global__ void copyOddsP(int*inp, int*prefix, int*inpLen,int*out){
    if((blockIdx.x+threadIdx.x)==0){ out[0]=inp[0];}
    else if((blockIdx.x+threadIdx.x)<*inpLen){
        int i=threadIdx.x + blockIdx.x*tpb;
        if(prefix[i]!=prefix[i-1]){
            out[prefix[i-1]]=inp[i];
        }
    }
}

int* filter(int* cudNum, int numLen, int bit, int value, int** zeroLen){
    int* cBit;
    int* cVal;
    hipMalloc(&cBit,sizeof(int));
    hipMalloc(&cVal,sizeof(int));
    hipMemcpy(cBit,&bit,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(cVal,&value,sizeof(int),hipMemcpyHostToDevice);
    int falseLen=1;
    while(falseLen<numLen) falseLen*=2;
    int Len=falseLen;
    int* cudLen;
    hipMalloc(&cudLen,sizeof(int));
    hipMemcpy(cudLen,&Len,sizeof(int),hipMemcpyHostToDevice);
    int* trueLen;
    hipMalloc(&trueLen,sizeof(int));
    hipMemcpy(trueLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    //printArr<<<1,1>>>(cudNum,trueLen);
    int* out;
    hipMalloc(&out,(Len+1)*sizeof(int));
    int* last;
    hipMalloc(&last,sizeof(int));
    bitMask<<<(Len+tpb)/tpb,tpb>>>(cudNum,cudLen,out,last,cBit,cVal);
    for(int step=1; step<Len; step*=2){ upSweep<<<(Len+tpb)/tpb,tpb>>>(out,cudLen,trueLen,step); }
    for(int step=Len/2; step>0; step/=2){ downSweep<<<(Len+tpb)/tpb,tpb>>>(out,cudLen,trueLen,step); }
    Len=numLen;
    cudLen=trueLen;
    int* shifted;
    hipMalloc(&shifted,Len*sizeof(int));
    exToIn<<<(Len+tpb)/tpb,tpb>>>(out,shifted,cudLen,last);
    int* cudOut;
    hipMalloc((void**) &cudOut, Len*sizeof(int));
    copyOddsP<<<(Len+tpb)/tpb,tpb>>>(cudNum, shifted, cudLen,cudOut);   
    *zeroLen = last;
    //cudaFree(cudLen);
    //cudaFree(cudNum);
    //cudaFree(out);
    //cudaFree(last);
    //cudaFree(shifted);
    return cudOut;
}

__global__ void copyArr(int*a, int*b, int*c, int* lenB, int*lenC){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(index>=((*lenB)+(*lenC))) return;
    if(index<(*lenB)) a[index]=b[index];
    else a[index]=c[index-(*lenB)];
}

int main(int argc,char **argv){
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    int* zerLen;
    int* oneLen;
    int* start;
    int* end; 
    int* cudLen;
    int maxBit=1;
    int* cudNum;
    hipMalloc(&cudNum,(numLen*sizeof(int)));
    hipMemcpy(cudNum,inp,(numLen)*sizeof(int),hipMemcpyHostToDevice);
    while((1<<maxBit)<numLen) maxBit++;
    hipMalloc(&cudLen,sizeof(int));
    hipMemcpy(cudLen,&numLen,sizeof(int),hipMemcpyHostToDevice); 
    for(int i=0; i<10; i++){
        start=filter(cudNum,numLen,i,0, &zerLen);
        end=filter(cudNum,numLen,i,1,&oneLen);
        copyArr<<<(numLen+tpb)/tpb,tpb>>>(cudNum,start,end,zerLen,oneLen);
    }
    hipMemcpy(inp,cudNum,numLen*sizeof(int),hipMemcpyDeviceToHost);
    for(int j=0; j<numLen; j++) printf("%d\n",inp[j]);
}
