
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define THREADS_PER_BLOCK 64

__global__ void part_a_cuda(int* a, int* b, int len)
{
    int a_index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int b_index = 0;
    
	if (a_index < len) {
		b_index = a[a_index] / 100;
		atomicAdd(&b[b_index], 1);
	}
}

__global__ void part_b_cuda(int* a, int* b, int len)
{
	int a_index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int b_index = 0;
    __shared__ int temp[10];
    
	if (a_index < len) {
		b_index = a[a_index] / 100;
		atomicAdd(&temp[b_index], 1);
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = 0; i < 10; i++) {
			atomicAdd(&b[i], temp[i]);
		}
	}
}

__global__ void part_c_cuda(int* a, int* b)
{
	//part c, prefix sum with 10 elements
}

void part_a() {
	//gather input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1, sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 

    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * 10);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    //cuda stuff
    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);
    part_a_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * 10, hipMemcpyDeviceToHost);

    //Print final values
    /*
    for(int i = 0; i < 10; i++) {
    	printf("%d, ", B[i]);
    }
    */
    fclose(fp);
	FILE* fp_end = fopen("q2a.txt", "w");
	for (int i = 0; i < 10; i++) {
		fputc(B[i] + '0', fp_end);
		if (i != 9) {
			fputc(', ', fp_end);
		}
	}
	fclose(fp_end);
	hipFree(d_a);
	hipFree(d_b);
}

void part_b() {
	//gather input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1, sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 

    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * 10);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    //cuda stuff
    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);
    part_b_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * 10, hipMemcpyDeviceToHost);

    //Print final values
    /*
    for(int i = 0; i < 10; i++) {
    	printf("%d, ", B[i]);
    }
    */
    fclose(fp);
	FILE* fp_end = fopen("q2b.txt", "w");
	for (int i = 0; i < 10; i++) {
		fputc(B[i] + '0', fp_end);
		if (i != 9) {
			fputc(', ', fp_end);
		}
	}
	fclose(fp_end);
	hipFree(d_a);
	hipFree(d_b);
}

void part_c() {
	//gather input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1, sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 

    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * 10);
    int* C = (int* )malloc(sizeof(int) * 10);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    //cuda stuff
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);
    part_b_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();

    fclose(fp);
    hipFree(d_a);

	//now do prefix sum of 10 elements in b
    hipMalloc(&d_c, sizeof(int) * 10);
    part_c_cuda<<<(10 + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_b, d_c);
    hipDeviceSynchronize();
    hipMemcpy(d_c, C, sizeof(int) * 10, hipMemcpyDeviceToHost);

    //copy stuff to file
	FILE* fp_end = fopen("q2c.txt", "w");
	for (int i = 0; i < 10; i++) {
		fputc(C[i] + '0', fp_end);
		if (i != 9) {
			fputc(', ', fp_end);
		}
	}
	fclose(fp_end);
	
}

int main(int argc, char **argv)
{
    part_a();
    part_b();
    part_c();
    return 0;
}