
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define THREADS_PER_BLOCK 64

__global__ void part_a_cuda(int* a, int* b, int len)
{
    int a_index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int b_index = 0;
    
	if (a_index < len) 
		b_index = a[a_index] / 100;
		b[b_index]++; 
}

void part_a() {
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1, sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 

    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * 10);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);
    part_a_cuda<<<len/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * 10, hipMemcpyDeviceToHost);

    for(int i = 0; i < 10; i++) {
    	printf("%d, ", B[i]);
    }

}

int main(int argc, char **argv)
{
    part_a();
    printf("That's all for part a!\n");
    //part_b()
    //printf("That's all for part b!\n");
    //part_c()
    //printf("That's all for part c!\n");

    return 0;
}