#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>

#define THREADS_PER_BLOCK 64

__global__ void part_a_cuda(int* a, int* b, int len)
{
    int a_index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int b_index = 0;
    
	if (a_index < len) {
		b_index = a[a_index] / 100;
		atomicAdd(&b[b_index], 1);
	}
}

void part_a() {
	//gather input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1, sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 

    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * 10);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    //cuda stuff
    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);
    part_a_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * 10, hipMemcpyDeviceToHost);

    //Print final values
    /*
    for(int i = 0; i < 10; i++) {
    	printf("%d, ", B[i]);
    }
    */
    fclose(fp);
	FILE* fp_end = fopen("q2a.txt", "w");
	for (int i = 0; i < 10; i++) {
		fputc(B[i] + '0', fp_end);
		if (i != 9) {
			fputc(', ', fp_end);
		}
	}
	fclose(fp_end);
	hipFree(d_a);
	hipFree(d_b);
}

int main(int argc, char **argv)
{
    part_a();
    part_b();
    part_c();
    return 0;
}