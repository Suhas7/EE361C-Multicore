
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define THREADS_PER_BLOCK 64

__global__ void part_a_cuda(int* a, int* b, int len)
{
    int a_index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int b_index = 0;
    
	if (a_index < len) {
		b_index = a[a_index] / 100;
		atomicAdd(&b[b_index], 1);
	}
}

__global__ void part_b_cuda(int* a, int* b, int len)
{
	int a_index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int b_index = 0;
    __shared__ int temp[10];
    
	if (a_index < len) {
		b_index = a[a_index] / 100;
		atomicAdd(&temp[b_index], 1);
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = 0; i < 10; i++) {
			atomicAdd(&b[i], temp[i]);
		}
	}
}

__global__ void part_c_cuda(int* a, int* b)
{
    b[threadIdx.x] = 0;
    for (int i = 0; i <= threadIdx.x; i++) {
        b[threadIdx.x] += a[i];
    }
}

void part_a() {
	//gather input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1, sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 

    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * 10);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }
    fclose(fp);

    //cuda stuff
    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);
    part_a_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * 10, hipMemcpyDeviceToHost);

    
	FILE* fp_end = fopen("q2a.txt", "w");
	for (int i = 0; i < 10; i++) {
		fprintf(fp_end, "%d", B[i]);
		if (i != 9) {
			fprintf(fp_end, "%s", ", ");
		}
	}
	fclose(fp_end);
	hipFree(d_a);
	hipFree(d_b);
    free(A);
    free(B);
}

void part_b() {
	//gather input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1, sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 

    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * 10);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }
    fclose(fp);

    //cuda stuff
    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);
    part_b_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * 10, hipMemcpyDeviceToHost);

	FILE* fp_end = fopen("q2b.txt", "w");
	for (int i = 0; i < 10; i++) {
		fprintf(fp_end, "%d", B[i]);
		if (i != 9) {
			fprintf(fp_end, "%s", ", ");
		}
	}

	fclose(fp_end);
	hipFree(d_a);
	hipFree(d_b);
    free(A);
    free(B);
}

void part_c() {
	//gather input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1, sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 

    while(token != NULL) {
        inp[len] = atoi(token+1);
        len++;
        token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * 10);

    for (int i = 0; i < len; i++) {
        A[i] = inp[i];
    }
    fclose(fp);

    //cuda stuff
    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);
    part_b_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * 10, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);

    int *d_c;
    int* C = (int* )malloc(sizeof(int) * 10);

	//now do prefix sum of 10 elements in b
    hipMalloc(&d_c, sizeof(int) * 10);
    hipMalloc(&d_b, sizeof(int) * 10);
    hipMemcpy(d_b, B, sizeof(int) * 10, hipMemcpyHostToDevice);
    part_c_cuda<<<1, 10>>>(d_b, d_c);
    hipDeviceSynchronize();
    hipMemcpy(C, d_c, sizeof(int) * 10, hipMemcpyDeviceToHost);

    //copy stuff to file
	FILE* fp_end = fopen("q2c.txt", "w");
	for (int i = 0; i < 10; i++) {
		fprintf(fp_end, "%d", C[i]);
		if (i != 9) {
			fprintf(fp_end, "%s", ", ");
		}
	}
	fclose(fp_end);
    hipFree(d_c);
    hipFree(d_b);	
    free(C);
    free(B);
    free(A);
}

int main(int argc, char **argv)
{
    part_a();
    hipDeviceReset();
    part_b();
    hipDeviceReset();
    part_c();

    return 0;
}