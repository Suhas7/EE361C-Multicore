
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define THREADS_PER_BLOCK 32

__global__ void minA_cuda(int* a, int* b, int len, int n_output) {
	int b_index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int a_index = b_index * 2;
    
    if (b_index < n_output) {
	    if (a_index == len) {
	    	b[b_index] = a[a_index];
	    }
	    else {
		    int v1 = a[a_index];
		    int v2 = a[a_index + 1];
		    if (v2 < v1) {
		    	b[b_index] = v2;
		    }
		    else {
		    	b[b_index] = v1;
		    }
		}
	}
}

__global__ void last_digit_cuda(int* a, int* b, int len) {
	int index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
	if (index < len) 
		b[index] = a[index] % 10;
}

void last_digit() {
	//Get input data from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1,sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 
    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * len);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    //Cuda stuff
    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * len);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);

    last_digit_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * len, hipMemcpyDeviceToHost);

    //put results in file
	fclose(fp);
	FILE* fp_end = fopen("q1b.txt", "w");
	for (int i = 0; i < len; i++) {
		fprintf(fp_end, "%d", B[i]);
		if (i != len-1) {
			fprintf(fp_end, "%s", ", ");
		}
	}

	//Free up memory
	hipFree(d_a);
	hipFree(d_b);
	free(A);
	free(B);
	fclose(fp_end);
}

void minA() {
	//Get input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1,sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 
    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    //Copy input to array of proper size
    int* A = (int* )malloc(sizeof(int) * len);
    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    //Cuda stuff
    int B_size = (len + 1) / 2;
    int* B;

    while (len != 1) {
    	B = (int* )malloc(sizeof(int) * B_size);

		int *d_a, *d_b;
	    hipMalloc(&d_a, sizeof(int) * len);
	    hipMalloc(&d_b, sizeof(int) * B_size);
	    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);

	    minA_cuda<<<(B_size + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len, B_size);
	    hipDeviceSynchronize();
	    hipMemcpy(B, d_b, sizeof(int) * B_size, hipMemcpyDeviceToHost);

	    hipFree(d_a);
	    hipFree(d_b);
	    memcpy(A, B, B_size * sizeof(int));
	    free(B);
	    len = B_size;
	    B_size = (len + 1) / 2;
    }

    //Print output to file
    fclose(fp);
	FILE* fp_end = fopen("q1a.txt", "w");
	fprintf(fp_end, "%d", A[0]);
	fclose(fp_end);
}

int main(int argc,char **argv)
{
    minA();
    hipDeviceReset();
    last_digit();
    return 0;
}
