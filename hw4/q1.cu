
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define THREADS_PER_BLOCK 32

__global__ void minA_cuda(int* a, int* b, int len) {
    //Min A
}

__global__ void last_digit_cuda(int* a, int* b, int len) {
	int index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
	if (index < len) 
		b[index] = a[index] % 10;
}

void last_digit() {
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1,sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 
    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * len);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * len);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);

    last_digit_cuda<<<len/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * len, hipMemcpyDeviceToHost);

    for(int i =0; i < len; i++) {
    	printf("%d, ", B[i]);
    }

	printf("That's all!\n");
}

void minA() {
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1,sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 
    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

}

int main(int argc,char **argv)
{
    // launch the kernel
    //hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    //cudaDeviceSynchronize();

    //printf("That's all!\n");
    last_digit();

    return 0;
}
