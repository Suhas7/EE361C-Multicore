
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define THREADS_PER_BLOCK 32

__global__ void minA_cuda(int* a, int* b, int len, int n_output) {
	int b_index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int a_index = b_index * 2;
    
    if (b_index < n_output) {
	    if (a_index == len) {
	    	b[b_index] = a[a_index];
	    }
	    else {
		    int v1 = a[a_index];
		    int v2 = a[a_index + 1];
		    if (v2 < v1) {
		    	b[b_index] = v2;
		    }
		    else {
		    	b[b_index] = v1;
		    }
		}
	}
}

__global__ void last_digit_cuda(int* a, int* b, int len) {
	int index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
	if (index < len) 
		b[index] = a[index] % 10;
}

void last_digit() {
	//Get input data from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1,sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 
    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    int* A = (int* )malloc(sizeof(int) * len);
    int* B = (int* )malloc(sizeof(int) * len);

    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    //Cuda stuff
    int *d_a, *d_b;
    hipMalloc(&d_a, sizeof(int) * len);
    hipMalloc(&d_b, sizeof(int) * len);
    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);

    last_digit_cuda<<<(len + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len);
    hipDeviceSynchronize();
    hipMemcpy(B, d_b, sizeof(int) * len, hipMemcpyDeviceToHost);

    //put results in file
	fclose(fp);
	FILE* fp_end = fopen("q1b.txt", "w");
	for (int i = 0; i < len; i++) {
		fputc(B[i] + '0', fp_end);
		if (i != len-1) {
			fputc(',', fp_end);
			fputc(' ', fp_end);
		}
	}

	//Free up memory
	hipFree(d_a);
	hipFree(d_b);
	free(A);
	free(B);
	fclose(fp_end);
}

void minA() {
	//Get input from files
    char buff[50000];
    int inp[10000];
    buff[0] = ' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r");
    fgets(buff+1,sizeof(buff), fp);
    token = strtok(buff, ",");
    int len = 0; 
    while(token != NULL) {
		inp[len] = atoi(token+1);
		len++;
		token = strtok(NULL, ",");
    }

    //Edge case where there is only one element
    if (len == 1) {
    	printf("%d", inp[0]);
    }

    //Copy input to array of proper size
    int* A = (int* )malloc(sizeof(int) * len);
    for (int i = 0; i < len; i++) {
    	A[i] = inp[i];
    }

    //Cuda stuff
    int B_size = (len + 1) / 2;
    int* B;

    while (B_size != 0) {
    	B = (int* )malloc(sizeof(int) * B_size);

		int *d_a, *d_b;
	    hipMalloc(&d_a, sizeof(int) * len);
	    hipMalloc(&d_b, sizeof(int) * B_size);
	    hipMemcpy(d_a, A, sizeof(int) * len, hipMemcpyHostToDevice);

	    minA_cuda<<<(B_size + THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, len, B_size);
	    hipDeviceSynchronize();
	    hipMemcpy(B, d_b, sizeof(int) * B_size, hipMemcpyDeviceToHost);

	    hipFree(d_a);
	    hipFree(d_b);
	    memcpy(A, B, B_size * sizeof(int));
	    free(B);
	    len = B_size;
	    B_size = (len + 1) / 2;
    }

    //Print output to file
    fclose(fp);
	FILE* fp_end = fopen("q1a.txt", "w");
	fputc(A[0] + '0', fp_end);
	fclose(fp_end);
}

int main(int argc,char **argv)
{
    last_digit();
    minA();
    return 0;
}
