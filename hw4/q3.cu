
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define  tpb 32

__global__ void oddCheck(int* nums,int*len, int* out, int* last){
    int index=threadIdx.x + blockIdx.x*tpb;
    if (index<*len) out[index]=nums[index]%2;
    if(index==((*len)-1)) *last=out[index];
}

__global__ void exToIn(int* inp, int* out, int*len, int*last){
    int index = threadIdx.x + blockIdx.x*tpb;
    if((index>0)&&(index<*len)){
        out[index-1]=inp[index];
    }
    if(index==((*len)-1)) { out[index]=inp[index]+*last;
    *last=out[index];
    }
}

__global__ void upSweep(int* arr, int* len, int* tLen, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(index>*tLen) return;
    if((((index+1)%(step*2))!=0) || index==0 || ((*len)<=index)) return;
    arr[index]=arr[index]+arr[index-step];
}

__global__ void downSweep(int* arr, int* len, int* tLen, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(2*step==*len) arr[(*len)-1]=0;
    if((((index+1)%(step*2))!=0) || (index==0) || ((*len)<=index)) return;    
    int tmp=arr[index-step];
    arr[index-step]=arr[index];
    arr[index]+=tmp;
}

__global__ void printArr(int* arr,int*len){
    for(int i=0;i<((*len)-1);i++) printf("%d, ",arr[i]);
    printf("%d",arr[(*len)-1]);
}

__global__ void copyOddsP(int*inp, int*prefix, int*inpLen,int*out){
    if((blockIdx.x+threadIdx.x)==0){ out[0]=inp[0];}
    else if((blockIdx.x+threadIdx.x)<*inpLen){
        int i=threadIdx.x + blockIdx.x*tpb;
        if(prefix[i]!=prefix[i-1]){
            out[prefix[i-1]]=inp[i];
        }
    }
}

int main(int argc,char **argv){
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    int* nums = inp;
    int falseLen=1;
    while(falseLen<numLen) falseLen*=2;
    int Len=falseLen;
    int* cudLen;
    hipMalloc(&cudLen,sizeof(int));
    hipMemcpy(cudLen,&Len,sizeof(int),hipMemcpyHostToDevice);
    int* trueLen;
    hipMalloc(&trueLen,sizeof(int));
    hipMemcpy(trueLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* cudNum;
    hipMalloc(&cudNum,(Len)*sizeof(int));
    hipMemcpy(cudNum,nums,(Len)*sizeof(int),hipMemcpyHostToDevice);
    int* out;
    hipMalloc(&out,(Len+1)*sizeof(int));
    int* last;
    hipMalloc(&last,sizeof(int));
    oddCheck<<<(Len+tpb)/tpb,tpb>>>(cudNum,trueLen,out,last);
    for(int step=1; step<Len; step*=2){
        upSweep<<<(Len+tpb)/tpb,tpb>>>(out,cudLen,trueLen,step);
    }
    for(int step=Len/2; step>0; step/=2){
        downSweep<<<(Len+tpb)/tpb,tpb>>>(out,cudLen,trueLen,step);
    }
    Len=numLen;
    cudLen=trueLen;
    int* shifted;
    hipMalloc(&shifted,Len*sizeof(int));
    exToIn<<<(Len+tpb)/tpb,tpb>>>(out,shifted,cudLen,last);
    int* cudOut;
    hipMalloc((void**) &cudOut, Len*sizeof(int));
    copyOddsP<<<(Len+tpb)/tpb,tpb>>>(cudNum, shifted, cudLen,cudOut); 
    int len;
    hipMemcpy(&len,last,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(inp,cudOut,len*sizeof(int),hipMemcpyDeviceToHost);
    fclose(fp);
    FILE* fp_end = fopen("q3.txt", "w");
    for (int i = 0; i < len; i++) {
        fprintf(fp_end, "%d", inp[i]);
        if (i != len-1) {
            fprintf(fp_end, ", ");
        }
    }
    hipFree(cudLen);
    hipFree(cudNum);
    hipFree(out);
    hipFree(last);
    hipFree(shifted);
    hipFree(cudOut);
    return 0;
}
