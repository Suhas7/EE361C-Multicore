
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define  tpb 32

__global__ void oddCheck(int* nums,int*len, int* out, int* last){
    int index=threadIdx.x + blockIdx.x*tpb;
    if (index<*len) out[index]=nums[index]%2;
    if(index==((*len)-1)) *last=out[index];
}

__global__ void exToIn(int* inp, int* out, int*len, int*last){
    int index = threadIdx.x + blockIdx.x*tpb;
    if((index>0)&&(index<*len)){
        out[index-1]=inp[index];
    }
    if(index==((*len)-1)) { out[index]=inp[index]+*last;
    *last=out[index];
    }
}

__global__ void upSweep(int* arr, int* len, int* tLen, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(index>*tLen) return;
    if((((index+1)%(step*2))!=0) || index==0 || ((*len)<=index)) return;
    arr[index]=arr[index]+arr[index-step];
}

__global__ void downSweep(int* arr, int* len, int* tLen, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(2*step==*len) arr[(*len)-1]=0;
    if((((index+1)%(step*2))!=0) || (index==0) || ((*len)<=index)) return;    
    int tmp=arr[index-step];
    arr[index-step]=arr[index];
    arr[index]+=tmp;
}

__global__ void printArr(int* arr,int*len){
    for(int i=0;i<((*len)-1);i++) printf("%d, ",arr[i]);
    printf("%d",arr[(*len)-1]);
}

__global__ void copyOddsP(int*inp, int*prefix, int*inpLen,int*out){
    if((blockIdx.x+threadIdx.x)==0){ out[0]=inp[0];}
    else if((blockIdx.x+threadIdx.x)<*inpLen){
        int i=threadIdx.x + blockIdx.x*tpb;
        if(prefix[i]!=prefix[i-1]){
            out[prefix[i-1]]=inp[i];
        }
    }
}

int main(int argc,char **argv){
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    int* nums = inp;
    int falseLen=1;
    while(falseLen<numLen) falseLen*=2;
    int Len=falseLen;
    int* cudLen;
    hipMalloc(&cudLen,sizeof(int));
    hipMemcpy(cudLen,&Len,sizeof(int),hipMemcpyHostToDevice);
    int* trueLen;
    hipMalloc(&trueLen,sizeof(int));
    hipMemcpy(trueLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* cudNum;
    hipMalloc(&cudNum,(Len)*sizeof(int));
    hipMemcpy(cudNum,nums,(Len)*sizeof(int),hipMemcpyHostToDevice);
    int* out;
    hipMalloc(&out,(Len+1)*sizeof(int));
    int* last;
    hipMalloc(&last,sizeof(int));
    oddCheck<<<(Len+tpb)/tpb,tpb>>>(cudNum,cudLen,out,last);
    for(int step=1; step<Len; step*=2){
        upSweep<<<(Len+tpb)/tpb,tpb>>>(out,cudLen,trueLen,step);
    }
    for(int step=Len/2; step>0; step/=2){
        downSweep<<<(Len+tpb)/tpb,tpb>>>(out,cudLen,trueLen,step);
    }
    Len=numLen;
    cudLen=trueLen;
    int* shifted;
    hipMalloc(&shifted,Len*sizeof(int));
    exToIn<<<(Len+tpb)/tpb,tpb>>>(out,shifted,cudLen,last);
    int* cudOut;
    hipMalloc((void**) &cudOut, Len*sizeof(int));
    copyOddsP<<<(Len+tpb)/tpb,tpb>>>(cudNum, shifted, cudLen,cudOut); 
    printArr<<<1,1>>>(cudOut,last);
    hipFree(cudLen);
    hipFree(cudNum);
    hipFree(out);
    hipFree(last);
    hipFree(shifted);
    hipFree(cudOut);
    return 0;
}
