#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#define NUM_BLOCKS 1
#define BLOCK_WIDTH 1

__global__ void kern(int* inp, int* inpLen, int* res, int* resLen){
    int j=0;
    for(int i = 0; i<*inpLen; i++){
        if(inp[i]%2==1){
            res[j]=inp[i];
            j++;
        }
    }
    *resLen=j;
}

__global__ void prefixSum(int* inp, int inpLen, int* res, int* resLen){
    int j=0;
    for(int i = 0; i<inpLen; i++){
        if(inp[i]%2==1){
            res[j]=inp[i];
            j++;
        }
    }
    *resLen=j;
}
__global__ void copyInt(int* dest, int* source){
    *dest=*source;
}
__global__ void copyOdds(int* inp, int* prefix, int* inpLen, int* out){
    if(prefix[0]==1) out[0]=inp[0];
    //todo parallelize this loop
    for(int i=1; i<inpLen; i++){
        if(prefix[i]!=prefix[i-1]) out[prefix[i]-1]=inp[i];
    }
}

__global__ void driver(int* cudaInp,int* inpLen,int** resArr, int* resLen){
    int* prefix;
    hipMalloc((void**)&prefix,inpLen*sizeof(int));
    //compute prefixSum
    prefixSum(inp, *inpLen, prefix, resLen);
    //allocate output array
    hipMalloc(*resArr, (*resLen)*sizeof(int));
    copyOdds(inp, prefix, inpLen, *resArr);
    hipFree(prefix);
}

int main(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[5000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    //GPU data transfer
    printf("started gpu sect\n");
    int* cudaInp;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMemcpy(cudaInp, inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    int* inpLen;
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* resLen;
    hipMalloc((void**)&resLen,sizeof(int));
    int** resArr;
    
    //run kernel
    kern<<<NUM_BLOCKS, BLOCK_WIDTH>>>(cudaInp,inpLen,resArr, resLen);
    hipDeviceSynchronize();
    
    //recover data
    int resLenHost;
    hipMemcpy(&resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    printf("Result is size %i\n",resLenHost);
    
    hipMemcpy(inp, resArr, resLenHost*sizeof(int), hipMemcpyDeviceToHost);
    for(int j=0; j<resLenHost;j++){
        printf("%d\n",inp[j]);
    }
    hipFree(cudaInp);
    hipFree(inpLen);
    hipFree(*resArr);
    hipFree(resLen);
}
