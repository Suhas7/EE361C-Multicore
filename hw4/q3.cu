
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define NUM_BLOCKS 1
#define BLOCK_WIDTH 1

//todo this needs parallelization
__device__ void prefixSum(int* inp, int* inpLen, int* res, int* resLen){
    int runningTotal=0;
    int length=*inpLen;
    for(int i=0; i<length;i++){
        if((inp[i]%2)==1) runningTotal++;
        res[i]=runningTotal;
    }
    *resLen=runningTotal;
}

__device__ void copyInt(int* dest, int* source){
    *dest=*source;
}

__device__ void copyOdds(int* inp, int* prefix, int* inpLen, int* out){
    if(prefix[0]==1) out[0]=inp[0];
    //todo parallelize this loop
    for(int i=1; i<*inpLen; i++){
        if(prefix[i]!=prefix[i-1]) inp[prefix[i]-1]=inp[i];
    }
}

__global__ void driver(int* cudaInp,int* inpLen,int** resArr, int* resLen){
    int* prefix = (int*) malloc((*inpLen)*sizeof(int));
    //compute prefixSum
    prefixSum(cudaInp, inpLen, prefix,resLen);
    //allocate output array
    *resArr = (int*) malloc((*resLen)*sizeof(int));
    //postprocess to make an array of odds
    *resLen=prefix[*inpLen-1];
    copyOdds(cudaInp, prefix, inpLen, *resArr);
    //print output
    for(int i=0; i<*resLen; i++){
        printf("%d\n",cudaInp[i]);
    }
    free(prefix);
}

int main(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    //GPU data transfer
    printf("started gpu sect\n");
    int* cudaInp;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMemcpy(cudaInp, inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    int* inpLen;
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* resLen;
    hipMalloc((void**)&resLen,sizeof(int));
    int** resArr;
    hipMalloc((void**)&resArr,sizeof(int*));
    
    //run kernel
    driver<<<NUM_BLOCKS, BLOCK_WIDTH>>>(cudaInp,inpLen,resArr, resLen);
    hipDeviceSynchronize();

    //recover data
    int resLenHost=7;
    hipMemcpy(&resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    printf("Result is size %i\n",resLenHost);
    /*
    cudaFree(cudaInp);
    cudaFree(inpLen);
    cudaFree(*resArr);
    cudaFree(resLen);
*/}
