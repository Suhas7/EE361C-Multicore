
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define NUM_BLOCKS 1
#define BLOCK_WIDTH 1

__global__ void kern(int* inp, int inpLen, int* res, int* resLen){
    int j=0;
    for(int i = 0; i<inpLen; i++){
        if(inp[i]%2==1){
            res[j]=inp[i];
            j++;
        }
    }
    *resLen=j;
}


int main(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[5000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }

    //transmit to GPU
    printf("started gpu sect\n");
    int* cudaInp;
    int* resArr;
    int resLen;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMalloc((void**)&resArr,numLen*sizeof(int));
    hipMemcpy(cudaInp, &inp, numLen*sizeof(int), hipMemcpyHostToDevice);

    //run kernel
    kern<<<NUM_BLOCKS, BLOCK_WIDTH>>>(cudaInp,numLen,resArr,&resLen);
    hipDeviceSynchronize();
    printf("%d\n",resLen);
    for(int j=0; j<resLen;j++){
        printf("%d\n",resArr[j]);
    }
    hipFree(cudaInp);
}
