
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define NUM_BLOCKS 1
#define BLOCK_WIDTH 1

__global__ void kern(int* inp, int* inpLen, int* res, int* resLen){
    int j=0;
    for(int i = 0; i<*inpLen; i++){
        if(inp[i]%2==1){
            res[j]=inp[i];
            j++;
        }
    }
    *resLen=j;
}


int main(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[5000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }

    //transmit to GPU
    printf("started gpu sect\n");
    int* cudaInp;
    int* inpLen;
    int* resArr;
    int* resLen;
    int* resLenHost;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMalloc((void**)&resArr,numLen*sizeof(int));
    hipMalloc((void**)&resLen,sizeof(int));
    hipMemcpy(cudaInp, &inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);

    //run kernel
    kern<<<NUM_BLOCKS, BLOCK_WIDTH>>>(cudaInp,inpLen,resArr, resLen);
    hipDeviceSynchronize();
    hipMemcpy(inp, resArr, numLen*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    printf("%d\n",*resLenHost);
    for(int j=0; j<*resLenHost;j++){
        printf("%d\n",inp[j]);
    }
    hipFree(cudaInp);
}
