
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define NUM_BLOCKS 1
#define BLOCK_WIDTH 1

__global__ void kern(int* inp, int* inpLen, int* res, int* resLen){
    int j=0;
    for(int i = 0; i<*inpLen; i++){
        if(inp[i]%2==1){
            res[j]=inp[i];
            j++;
        }
    }
    *resLen=j;
}


int main(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[5000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    //transmit to GPU
    printf("started gpu sect\n");
    int* cudaInp;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    int* inpLen;
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMemcpy(cudaInp, inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    int* resArr;
    hipMalloc((void**)&resArr,numLen*sizeof(int));
    int* resLen;
    hipMalloc((void**)&resLen,sizeof(int));
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    
    //run kernel
    kern<<<NUM_BLOCKS, BLOCK_WIDTH>>>(cudaInp,inpLen,resArr, resLen);
    hipDeviceSynchronize();
    int resLenHost;
    hipMemcpy(&resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    printf("%i\n",resLenHost);
    hipMemcpy(inp, resArr, numLen*sizeof(int), hipMemcpyDeviceToHost);
    for(int j=0; j<resLenHost;j++){
        printf("%d\n",inp[j]);
    }
    hipFree(cudaInp);
}
