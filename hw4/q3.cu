
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#define  tpb 32

__global__ void oddCheck(int* nums,int*len, int* out, int* last){
    int index=threadIdx.x + blockIdx.x*tpb;
    if (index<*len) out[index]=nums[index]%2;
    if(index==((*len)-1)) *last=nums[index]+*last;
}

__global__ void exToIn(int* inp, int* out, int*len, int*last){
    int index=threadIdx.x + blockIdx.x*tpb;
    if((index>0)&&(index<*len)){
        out[index-1]=inp[index];
    }
    if(index==((*len)-1)) out[index]=inp[index]+*last;
}

__global__ void upSweep(int* arr, int* len, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if((((index+1)%(step*2))!=0) || index==0 || ((*len)<=index)) return;
    arr[index]=arr[index]+arr[index-step];
}

__global__ void downSweep(int* arr, int* len, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(2*step==*len) arr[(*len)-1]=0;
    if((((index+1)%(step*2))!=0) || (index==0) || ((*len)<=index)) return;
    int tmp=arr[index-step];
    arr[index-step]=arr[index];
    arr[index]+=tmp;
}

__global__ void printArr(int* arr,int*len){
    for(int i=0;i<(*len);i++) printf("%d\n",arr[i]);
}

void prefixSumP(int* inp, int* inpLen, int* res, int* resLen){
    //oddCheck<<<((*inpLen)+tpb)/tpb,tpb>>>(inp,inpLen,res);
    for(int step=1; step<*inpLen; step*=2){
        //upSweep<<<((*inpLen)+tpb)/tpb,tpb>>>(res,inpLen,step);
        upSweep<<<1,8>>>(res,inpLen,step);
    }
    //res[(*inpLen)-1]=0;
    for(int step=(*inpLen)/2; step>0; step/=2){
        //downSweep<<<((*inpLen)+tpb)/tpb,tpb>>>(res,inpLen,step);
        downSweep<<<1,8>>>(res,inpLen,step);
    }
    printArr<<<1,1>>>(res,inpLen);
    //*resLen=res[(*inpLen)-1]+(inp[(*inpLen)-1]%2);
}

__global__ void prefixSum(int* inp, int* inpLen, int* res, int* resLen){
    int runningTotal=0;
    int length=*inpLen;
    for(int i=0; i<length;i++){
        if((inp[i]%2)==1) runningTotal++;
        res[i]=runningTotal;
    }
    *resLen=runningTotal;
}

__global__ void copyOddsP(int*inp, int*prefix, int*inpLen,int*out){
    if((blockIdx.x+threadIdx.x)==0){ out[0]=inp[0];}
    else if((blockIdx.x+threadIdx.x)<*inpLen){
        int i=threadIdx.x + blockIdx.x*tpb;
        if(prefix[i]!=prefix[i-1]){
            out[prefix[i-1]]=inp[i];
        }
    }
}

void driver(int* cudaInp,int inpLen,int* cudInpLen, int* resLen){
    int* prefix;
    hipMalloc((void**)&prefix, inpLen*sizeof(int));
    //compute prefixSum
    prefixSum<<<1,1>>>(cudaInp, cudInpLen, prefix,resLen);
    //alloc
    int outLen;
    hipMemcpy(&outLen,resLen,sizeof(int),hipMemcpyDeviceToHost);
    int* cudOut;
    hipMalloc((void**) &cudOut, outLen*sizeof(int));
    //postprocess to make an array of odds
    copyOddsP<<<(inpLen+tpb)/tpb,tpb>>>(cudaInp, prefix, cudInpLen,cudOut);
    //print output
    int out[outLen];
    hipMemcpy(out,cudOut, outLen*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0; i<outLen; i++){
        printf("%d\n",out[i]);
    }
    hipFree(prefix);
}

int main2(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    //GPU data transfer
    int* cudaInp;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMemcpy(cudaInp, inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    int* inpLen;
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* resLen;
    hipMalloc((void**)&resLen,sizeof(int));
    
    //run kernel
    driver(cudaInp,numLen, inpLen, resLen);
    hipDeviceSynchronize();

    //recover data
    int resLenHost=7;
    hipMemcpy(&resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    
    hipFree(cudaInp);
    hipFree(inpLen);
    hipFree(resLen);
    return 0;
}

int main(int argc,char **argv){
    int nums[10]= {1,2,3,4,5,6,7,8,9,10};
    int Len=8;
    int* cudLen;
    hipMalloc(&cudLen,sizeof(int));
    hipMemcpy(cudLen,&Len,sizeof(int),hipMemcpyHostToDevice);
    int* cudNum;
    hipMalloc(&cudNum,(Len)*sizeof(int));
    hipMemcpy(cudNum,nums,(Len)*sizeof(int),hipMemcpyHostToDevice);
    int* out;
    hipMalloc(&out,(Len+1)*sizeof(int));
    int* last;
    hipMalloc(&last,sizeof(int));

    oddCheck<<<(Len+tpb)/tpb,tpb>>>(cudNum,cudLen,out,last);
    for(int step=1; step<8; step*=2){
        upSweep<<<(Len+tpb)/tpb,tpb>>>(out,cudLen,step);
    }
    //res[(*inpLen)-1]=0;
    for(int step=Len/2; step>0; step/=2){
        downSweep<<<(Len+tpb)/tpb,tpb>>>(out,cudLen,step);
    }
    int* shifted;
    hipMalloc(&shifted,Len*sizeof(int));
    exToIn<<<(Len+tpb)/tpb,tpb>>>(out,shifted,cudLen,last);
    printArr<<<1,1>>>(shifted,cudLen);
    //prefixSumP(cudNum, cudLen, out, cudLen);
    int* cudOut;
    hipMalloc((void**) &cudOut, Len*sizeof(int));
    //postprocess to make an array of odds
    copyOddsP<<<(Len+tpb)/tpb,tpb>>>(cudNum, shifted, cudLen,cudOut); 
    printArr<<<1,1>>>(cudOut,cudLen);
    hipFree(cudLen);
    hipFree(cudNum);
    return 0;
}
