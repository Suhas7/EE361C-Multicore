#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#define  tpb 32
//todo parallelize
__global__ void upSweep(int* arr, int* len, int* step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(((index+1)%(step*2)!=0) || index==0) return;
    arr[index]=arr[index]+arr[index-step];
}
__global__ void downSweep(int* arr, int* len, int* step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(((index+1)%(step*2)!=0) || index==0) return;
    arr[index]=arr[index]+arr[index-step];
}

__global__ void prefixSum(int* inp, int* inpLen, int* res, int* resLen){
    int runningTotal=0;
    int length=*inpLen;
    for(int i=0; i<length;i++){
        if((inp[i]%2)==1) runningTotal++;
        res[i]=runningTotal;
    }:
    *resLen=runningTotal;
}

__global__ void copyOddsP(int*inp, int*prefix, int*inpLen,int*out){
    if((blockIdx.x+threadIdx.x)==0){ out[0]=inp[0];}
    else if((blockIdx.x+threadIdx.x)<*inpLen){
        int i=threadIdx.x + blockIdx.x*tpb;
        if(prefix[i]!=prefix[i-1]){
            out[prefix[i-1]]=inp[i];
        }
    }
}

void driver(int* cudaInp,int inpLen,int* cudInpLen, int* resLen){
    int* prefix;
    hipMalloc((void**)&prefix, inpLen*sizeof(int));
    //compute prefixSum
    prefixSum<<<1,1>>>(cudaInp, cudInpLen, prefix,resLen);
    //alloc
    int outLen;
    hipMemcpy(&outLen,resLen,sizeof(int),hipMemcpyDeviceToHost);
    int* cudOut;
    hipMalloc((void**) &cudOut, outLen*sizeof(int));
    //postprocess to make an array of odds
    copyOddsP<<<(inpLen+tpb)/tpb,tpb>>>(cudaInp, prefix, cudInpLen,cudOut);
    //print output
    int out[outLen];
    hipMemcpy(out,cudOut, outLen*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0; i<outLen; i++){
        printf("%d\n",out[i]);
    }
    hipFree(prefix);
}

int main(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    //GPU data transfer
    int* cudaInp;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMemcpy(cudaInp, inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    int* inpLen;
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* resLen;
    hipMalloc((void**)&resLen,sizeof(int));
    
    //run kernel
    driver(cudaInp,numLen, inpLen, resLen);
    hipDeviceSynchronize();

    //recover data
    int resLenHost=7;
    hipMemcpy(&resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    
    hipFree(cudaInp);
    hipFree(inpLen);
    hipFree(resLen);
}
