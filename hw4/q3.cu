#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#define  tpb 32

__global__ void oddCheck(int* nums,int*len, int* out){
    int index=threadIdx.x + blockIdx.x*tpb;
    if (index<*len){ out[index]=nums[index]%2; }
}

//todo validate this
__global__ void upSweep(int* arr, int* len, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if((((index+1)%(step*2))!=0) || index==0) return;
    arr[index]=arr[index]+arr[index-step];
}

//todo validate this
__global__ void downSweep(int* arr, int* len, int step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if((((index+1)%(step*2))!=0) || index==0) return;
    int tmp=arr[index-step];
    arr[index-step]=arr[index];
    arr[index]+=tmp;
}

__global__ void printArr(int* arr,int*len){
    for(int i=0;i<(*len);i++) printf("%d",arr[i]);
}

void prefixSumP(int* inp, int* inpLen, int* res, int* resLen){
    oddCheck<<<((*inpLen)+tpb)/tpb,tpb>>>(inp,inpLen,res);
    for(int step=1; step<*inpLen; step*=2){
        upSweep<<<((*inpLen)+tpb)/tpb,tpb>>>(res,inpLen,step);
    }
    //res[(*inpLen)-1]=0;
    for(int step=*inpLen; step>0; step/=2){
        downSweep<<<((*inpLen)+tpb)/tpb,tpb>>>(res,inpLen,step);
    }
    printf("kernel print start");
    printArr<<<1,1>>>(res,inpLen);
    printf("kernel print end");
    *resLen=res[(*inpLen)-1]+(inp[(*inpLen)-1]%2);
}

__global__ void prefixSum(int* inp, int* inpLen, int* res, int* resLen){
    int runningTotal=0;
    int length=*inpLen;
    for(int i=0; i<length;i++){
        if((inp[i]%2)==1) runningTotal++;
        res[i]=runningTotal;
    }
    *resLen=runningTotal;
}

__global__ void copyOddsP(int*inp, int*prefix, int*inpLen,int*out){
    if((blockIdx.x+threadIdx.x)==0){ out[0]=inp[0];}
    else if((blockIdx.x+threadIdx.x)<*inpLen){
        int i=threadIdx.x + blockIdx.x*tpb;
        if(prefix[i]!=prefix[i-1]){
            out[prefix[i-1]]=inp[i];
        }
    }
}

void driver(int* cudaInp,int inpLen,int* cudInpLen, int* resLen){
    int* prefix;
    hipMalloc((void**)&prefix, inpLen*sizeof(int));
    //compute prefixSum
    prefixSum<<<1,1>>>(cudaInp, cudInpLen, prefix,resLen);
    //alloc
    int outLen;
    hipMemcpy(&outLen,resLen,sizeof(int),hipMemcpyDeviceToHost);
    int* cudOut;
    hipMalloc((void**) &cudOut, outLen*sizeof(int));
    //postprocess to make an array of odds
    copyOddsP<<<(inpLen+tpb)/tpb,tpb>>>(cudaInp, prefix, cudInpLen,cudOut);
    //print output
    int out[outLen];
    hipMemcpy(out,cudOut, outLen*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0; i<outLen; i++){
        printf("%d\n",out[i]);
    }
    hipFree(prefix);
}

int main2(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    //GPU data transfer
    int* cudaInp;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMemcpy(cudaInp, inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    int* inpLen;
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* resLen;
    hipMalloc((void**)&resLen,sizeof(int));
    
    //run kernel
    driver(cudaInp,numLen, inpLen, resLen);
    hipDeviceSynchronize();

    //recover data
    int resLenHost=7;
    hipMemcpy(&resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    
    hipFree(cudaInp);
    hipFree(inpLen);
    hipFree(resLen);
}

int main(int argc,char **argv){
    int nums[8]= {1,2,3,4,5,6,7,8};
    int Len=8;
    int* cudLen;
    hipMalloc(&cudLen,sizeof(int));
    hipMemcpy(cudLen,Len,sizeof(int),hipMemcpyHostToDevice);
    int* cudNum;
    hipMalloc(&cudNum,8*sizeof(int));
    hipMemcpy(cudNum,nums,8*sizeof(int),hipMemcpyHostToDevice);

    upSweep<<<2,4>>>(cudNum, cudLen, 1);
    printArr(cudNum,cudLen);
}