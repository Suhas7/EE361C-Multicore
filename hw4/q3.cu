
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

//todo parallelize
__device__ void prefixSum(int* inp, int* inpLen, int* res, int* resLen){
    int runningTotal=0;
    int length=*inpLen;
    for(int i=0; i<length;i++){
        if((inp[i]%2)==1) runningTotal++;
        res[i]=runningTotal;
    }
    *resLen=runningTotal;
}

__device__ void copyOdds(int* inp, int* prefix, int* inpLen){
    if(prefix[0]==1) inp[0]=inp[0];
    //todo parallelize
    for(int i=1; i<*inpLen; i++){
        if(prefix[i]!=prefix[i-1]) inp[prefix[i]-1]=inp[i];
    }
}

__global__ void driver(int* cudaInp,int* inpLen, int* resLen){
    int* prefix;
    //t index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    if(threadIdx.x==0){
        prefix = (int*) malloc((*inpLen)*sizeof(int));
        //compute prefixSum
        prefixSum(cudaInp, inpLen, prefix,resLen);
        //postprocess to make an array of odds
        *resLen=prefix[*inpLen-1];
    }
    __syncthreads();
    if(threadIdx.x==0){
    copyOdds(cudaInp, prefix, inpLen);
    //print output
    for(int i=0; i<*resLen; i++){
        printf("%d\n",cudaInp[i]);
    }
    free(prefix);
    }
    
}

int main(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    //GPU data transfer
    printf("started gpu sect\n");
    int* cudaInp;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMemcpy(cudaInp, inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    int* inpLen;
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* resLen;
    hipMalloc((void**)&resLen,sizeof(int));
    
    //run kernel
    driver<<<1,2>>>(cudaInp,inpLen, resLen);
    hipDeviceSynchronize();

    //recover data
    int resLenHost=7;
    hipMemcpy(&resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    printf("Result is size %i\n",resLenHost);
    
    hipFree(cudaInp);
    hipFree(inpLen);
    hipFree(resLen);
}
