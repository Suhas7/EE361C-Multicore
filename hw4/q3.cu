#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#define  tpb 32

__global__ void oddCheck(int* nums,int*len, int* out){
    int index=threadIdx.x + blockIdx.x*tpb;
    if (index<*len){ out[index]=nums[index]%2; }
}

//todo check this
__global__ void upSweep(int* arr, int* len, int* step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(((index+1)%(step*2)!=0) || index==0) return;
    arr[index]=arr[index]+arr[index-step];
}

//todo finish this
__global__ void downSweep(int* arr, int* len, int* step){
    int index=threadIdx.x + blockIdx.x*tpb;
    if(((index+1)%(step*2)!=0) || index==0) return;
    arr[index]=arr[index]+arr[index-step];
}

void prefixSumP(int* inp, int* inpLen, int* res, int* resLen){
    oddCheck<<<(inpLen+tpb)/tpb,tpb>>>(inp,inpLen,res);
    for(int step=1; step<*inpLen; step*=2){
        upSweep<<<(inpLen+tpb)/tpb,tpb>>>(res,inpLen,step);
    }
    for(int step=inpLen; step>0; step/=2){
        downSweep<<<(inpLen+tpb)/tpb,tpb>>>(res,inpLen,step);
    }
    *resLen=res[(*inpLen)-1];
}

__global__ void prefixSum(int* inp, int* inpLen, int* res, int* resLen){
    int runningTotal=0;
    int length=*inpLen;
    for(int i=0; i<length;i++){
        if((inp[i]%2)==1) runningTotal++;
        res[i]=runningTotal;
    }:
    *resLen=runningTotal;
}



__global__ void copyOddsP(int*inp, int*prefix, int*inpLen,int*out){
    if((blockIdx.x+threadIdx.x)==0){ out[0]=inp[0];}
    else if((blockIdx.x+threadIdx.x)<*inpLen){
        int i=threadIdx.x + blockIdx.x*tpb;
        if(prefix[i]!=prefix[i-1]){
            out[prefix[i-1]]=inp[i];
        }
    }
}

void driver(int* cudaInp,int inpLen,int* cudInpLen, int* resLen){
    int* prefix;
    hipMalloc((void**)&prefix, inpLen*sizeof(int));
    //compute prefixSum
    prefixSum<<<1,1>>>(cudaInp, cudInpLen, prefix,resLen);
    //alloc
    int outLen;
    hipMemcpy(&outLen,resLen,sizeof(int),hipMemcpyDeviceToHost);
    int* cudOut;
    hipMalloc((void**) &cudOut, outLen*sizeof(int));
    //postprocess to make an array of odds
    copyOddsP<<<(inpLen+tpb)/tpb,tpb>>>(cudaInp, prefix, cudInpLen,cudOut);
    //print output
    int out[outLen];
    hipMemcpy(out,cudOut, outLen*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0; i<outLen; i++){
        printf("%d\n",out[i]);
    }
    hipFree(prefix);
}

int main(int argc,char **argv){
    //read array in
    char buff[50000];
    int inp[15000];
    buff[0]=' ';
    char* token;
    FILE* fp = fopen("inp.txt", "r" );
    fgets(buff+1, 50000, fp);
    token=strtok(buff,",");
    int numLen=0;
    while(token!=NULL){
        inp[numLen]=atoi(token+1);
        numLen++;
        token=strtok(NULL,",");
    }
    //GPU data transfer
    int* cudaInp;
    hipMalloc((void**)&cudaInp,numLen*sizeof(int));
    hipMemcpy(cudaInp, inp, numLen*sizeof(int), hipMemcpyHostToDevice);
    int* inpLen;
    hipMalloc((void**)&inpLen,sizeof(int));
    hipMemcpy(inpLen,&numLen,sizeof(int),hipMemcpyHostToDevice);
    int* resLen;
    hipMalloc((void**)&resLen,sizeof(int));
    
    //run kernel
    driver(cudaInp,numLen, inpLen, resLen);
    hipDeviceSynchronize();

    //recover data
    int resLenHost=7;
    hipMemcpy(&resLenHost,resLen,sizeof(int),hipMemcpyDeviceToHost);
    
    hipFree(cudaInp);
    hipFree(inpLen);
    hipFree(resLen);
}
